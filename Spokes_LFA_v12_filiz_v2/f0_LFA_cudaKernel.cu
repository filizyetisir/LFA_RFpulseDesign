



#include <hip/hip_runtime.h>
#include <cmath>


#define TWO_PI_GAMMA (2.675128976384781e+08)




__device__ void multiply_2spinors(double a1_re,double a1_im,double b1_re,double b1_im,
				  double a2_re,double a2_im,double b2_re,double b2_im,
				  double *a3_re,double *a3_im,double *b3_re,double *b3_im)
{
  *a3_re = a2_re*a1_re - a2_im*a1_im - b2_re*b1_re - b2_im*b1_im;
  *a3_im = a2_im*a1_re + a2_re*a1_im + b2_im*b1_re - b2_re*b1_im;
  
  *b3_re = b2_re*a1_re - b2_im*a1_im + a2_re*b1_re + a2_im*b1_im;
  *b3_im = b2_im*a1_re + b2_re*a1_im - a2_im*b1_re + a2_re*b1_im;
}




__device__ void multiply_3spinors(double a1_re,double a1_im,double b1_re,double b1_im,
				  double a2_re,double a2_im,double b2_re,double b2_im,
				  double a3_re,double a3_im,double b3_re,double b3_im,
				  double *a4_re,double *a4_im,double *b4_re,double *b4_im)
{
  
  double tmp1_re = a2_re*a1_re - a2_im*a1_im - b2_re*b1_re - b2_im*b1_im;
  double tmp1_im = a2_im*a1_re + a2_re*a1_im + b2_im*b1_re - b2_re*b1_im;
  
  double tmp2_re = b2_re*a1_re - b2_im*a1_im + a2_re*b1_re + a2_im*b1_im;
  double tmp2_im = b2_im*a1_re + b2_re*a1_im - a2_im*b1_re + a2_re*b1_im;
  
  *a4_re = a3_re*tmp1_re - a3_im*tmp1_im - b3_re*tmp2_re - b3_im*tmp2_im;
  *a4_im = a3_im*tmp1_re + a3_re*tmp1_im + b3_im*tmp2_re - b3_re*tmp2_im;
  
  *b4_re = b3_re*tmp1_re - b3_im*tmp1_im + a3_re*tmp2_re + a3_im*tmp2_im;
  *b4_im = b3_im*tmp1_re + b3_re*tmp1_im - a3_im*tmp2_re + a3_re*tmp2_im;    
}




// CUDA kernel
__global__ void f0_LFA_cudaKernel(double *a_re,double *a_im,double *b_re,double *b_im,
				  double *datot_dre_re,double *datot_dre_im,double *datot_dim_re,double *datot_dim_im,
				  double *dbtot_dre_re,double *dbtot_dre_im,double *dbtot_dim_re,double *dbtot_dim_im,
				  double *da_dre_re,double *da_dre_im,double *da_dim_re,double *da_dim_im,
				  double *db_dre_re,double *db_dre_im,double *db_dim_re,double *db_dim_im,
				  double *a_forw_re,double *a_forw_im,double *b_forw_re,double *b_forw_im,
				  double *as_re,double *as_im,double *bs_re,double *bs_im,
				  double *b1s_re,double *b1s_im,double *grad,
				  double *time_to_sinc_time,double *time_to_spoke,double *subpulse,
				  double *b1maps_re,double *b1maps_im,
				  int nnzp,int ntimes,int ncoils,int nspokes,double deltat,int comp_grad)
{


  int index=threadIdx.x + blockIdx.x * blockDim.x;  //  global thread index == voxel index


  if(index<nnzp){

    
    // FORWARD BLOCH SIMULATION
    
    for(int time=0;time<ntimes;time++){
      
      int sinctime=(int)( time_to_sinc_time[time] );      
      
      double norm=pow(b1s_re[time*nnzp+index],2) + pow(b1s_im[time*nnzp+index],2) + pow(abs(grad[time*nnzp+index]),2);
      norm=sqrt(norm);
      
      double a2_re,a2_im,b2_re,b2_im;
      double n1,n2,n3,phi,cosphi,sinphi;
      if(norm==0){
	
	a2_re=1.0;
	a2_im=0.0;
	
	b2_re=0.0;
	b2_im=0.0;	
	
      }else{
	
	phi=-deltat*norm*TWO_PI_GAMMA;
	
	n1=b1s_re[time*nnzp+index] / norm;
	n2=b1s_im[time*nnzp+index] / norm;
	n3=grad[time*nnzp+index] / norm;    
	
	// new CK parameters
	cosphi=cos(phi/2.0);
	sinphi=sin(phi/2.0);
	
	a2_re=cosphi;
	a2_im=-n3*sinphi;
	
	b2_re=n2*sinphi;              
	b2_re=-n1*sinphi;              
	
	// forward Q product
	multiply_2spinors( a_re[index],a_im[index],b_re[index],b_im[index],
			   a2_re,a2_im,b2_re,b2_im,
			   &a_re[index],&a_im[index],&b_re[index],&b_im[index]);
      }
      
      
      if( comp_grad==1 ){
	
	// store Q matrix
	as_re[index + time*nnzp] = a2_re;
	as_im[index + time*nnzp] = a2_im;
	
	bs_re[index + time*nnzp] = b2_re;
	bs_im[index + time*nnzp] = b2_im;
	
	// store one time late forward Q matrix product
	if( time<ntimes-1 ){
	  a_forw_re[index + (time+1)*nnzp]=a_re[index];  // better way to handle memory?
	  a_forw_im[index + (time+1)*nnzp]=a_im[index];  
	  
	  b_forw_re[index + (time+1)*nnzp]=b_re[index];  
	  b_forw_im[index + (time+1)*nnzp]=b_im[index];  
	}
	
	if( sinctime>0 ){
	  
	  for(int j=0;j<ncoils;j++){
	    
	    if(norm>0){                    
	      
	      // derivative of phi wrt spokes amplitudes
	      double dphi_dre = -deltat/norm*subpulse[sinctime-1]*TWO_PI_GAMMA * ( b1s_re[index+time*nnzp]*b1maps_re[index+j*nnzp] + b1s_im[index+time*nnzp]*b1maps_im[index+j*nnzp] );
	      double dphi_dim = deltat/norm*subpulse[sinctime-1]*TWO_PI_GAMMA * ( b1s_re[index+time*nnzp]*b1maps_im[index+j*nnzp] - b1s_im[index+time*nnzp]*b1maps_re[index+j*nnzp] );
	      
	      // derivative of n (rotation axis) wrt spokes amplitudes
	      double tmp_re=-subpulse[sinctime-1]/pow(norm,3) * ( b1s_re[index+time*nnzp]*b1maps_re[index+j*nnzp] + b1s_im[index+time*nnzp]*b1maps_im[index+j*nnzp] );
	      double tmp_im=-subpulse[sinctime-1]/pow(norm,3) * ( b1s_re[index+time*nnzp]*b1maps_im[index+j*nnzp] - b1s_im[index+time*nnzp]*b1maps_re[index+j*nnzp] );
	      
	      double tmp3=subpulse[sinctime-1]/norm;            
	      
	      double dnx_dre=tmp_re*b1s_re[index+time*nnzp]  + tmp3*b1maps_re[index+j*nnzp];
	      double dnx_dim=-tmp_im*b1s_re[index+time*nnzp] - tmp3*b1maps_im[index+j*nnzp];
	      
	      double dny_dre=tmp_re*b1s_im[index+time*nnzp] + tmp3*b1maps_im[index+j*nnzp];
	      double dny_dim=-tmp_im*b1s_im[index+time*nnzp] + tmp3*b1maps_re[index+j*nnzp];
	      
	      double dnz_dre=tmp_re * grad[index+time*nnzp];
	      double dnz_dim=-tmp_im * grad[index+time*nnzp];
	      
	      // derivative of Q 
	      da_dre_re[index + time*nnzp + j*ntimes*nnzp]=-dphi_dre/2.0*sinphi;
	      da_dre_im[index + time*nnzp + j*ntimes*nnzp]=-dphi_dre/2.0*n3*cosphi - sinphi*dnz_dre;
	      
	      da_dim_re[index + time*nnzp + j*ntimes*nnzp]=-dphi_dim/2.0*sinphi;
	      da_dim_im[index + time*nnzp + j*ntimes*nnzp]=-dphi_dim/2.0*n3*cosphi - sinphi*dnz_dim;            
	      
	      db_dre_re[index + time*nnzp + j*ntimes*nnzp]= dny_dre*sinphi + 0.5*n2*cosphi*dphi_dre;
	      db_dre_im[index + time*nnzp + j*ntimes*nnzp]=-dnx_dre*sinphi - 0.5*n1*cosphi*dphi_dre;                                
	      
	      db_dim_re[index + time*nnzp + j*ntimes*nnzp]= dny_dim*sinphi + 0.5*n2*cosphi*dphi_dim;
	      db_dim_im[index + time*nnzp + j*ntimes*nnzp]=-dnx_dim*sinphi - 0.5*n1*cosphi*dphi_dim;
	      
	    }	  
	    
	  }  // for(j=0;j<ncoils;j++)
	  
	}  // if(sinctime>0)
	
      }  // if(comp_grad==1)            
      
    }  // for(time=0;times<ntimes;time++)     
    
    
    // BACKWARD BLOCH SIMULATION        
    if(comp_grad==1){
      
      a_re[index]=1.0;
      a_im[index]=0.0;
      
      b_re[index]=0.0;
      b_im[index]=0.0;
      
      for(int time=ntimes-1;time>=0;time--){            
	
	int sp_num=(int)( time_to_spoke[time] );

	double tmp3_re,tmp3_im,tmp4_re,tmp4_im;
	if(sp_num>0){               
	  
	  for(int j=0;j<ncoils;j++){
	    
	    // update derivatives of total Q matrices (sandwitch product)
	    multiply_3spinors( a_forw_re[index+time*nnzp],a_forw_im[index+time*nnzp],b_forw_re[index+time*nnzp],b_forw_im[index+time*nnzp],
			       da_dre_re[index + time*nnzp + j*ntimes*nnzp],da_dre_im[index + time*nnzp + j*ntimes*nnzp],db_dre_re[index + time*nnzp + j*ntimes*nnzp],db_dre_im[index + time*nnzp + j*ntimes*nnzp],
			       a_re[index],a_im[index],b_re[index],b_im[index],
			       &tmp3_re,&tmp3_im,&tmp4_re,&tmp4_im);                           
	    
	    datot_dre_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils]=datot_dre_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils] + tmp3_re;
	    datot_dre_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils]=datot_dre_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils] + tmp3_im;
	    
	    dbtot_dre_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils]=dbtot_dre_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils] + tmp4_re;
	    dbtot_dre_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils]=dbtot_dre_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils] + tmp4_im;
            
	    multiply_3spinors( a_forw_re[index+time*nnzp],a_forw_im[index+time*nnzp],b_forw_re[index+time*nnzp],b_forw_im[index+time*nnzp],
			       da_dim_re[index + time*nnzp + j*ntimes*nnzp],da_dim_im[index + time*nnzp + j*ntimes*nnzp],db_dim_re[index + time*nnzp + j*ntimes*nnzp],db_dim_im[index + time*nnzp + j*ntimes*nnzp],
			       a_re[index],a_im[index],b_re[index],b_im[index],
			       &tmp3_re,&tmp3_im,&tmp4_re,&tmp4_im);                           
	    
	    datot_dim_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils]=datot_dim_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils] + tmp3_re;
	    datot_dim_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils]=datot_dim_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils] + tmp3_im;
	    
	    dbtot_dim_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils]=dbtot_dim_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils] + tmp4_re;            
	    dbtot_dim_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils]=dbtot_dim_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils] + tmp4_im;
	    
	  }  // for(j=0;j<ncoils;j++)
          
	}  // if(sp_num>0)
	
	// keep track of the backward Q matrices product
	multiply_2spinors( as_re[index+time*nnzp],as_im[index+time*nnzp],bs_re[index+time*nnzp],bs_im[index+time*nnzp],
			   a_re[index],a_im[index],b_re[index],b_im[index],
			   &a_re[index],&a_im[index],&b_re[index],&b_im[index]);
        
      }  // for(time=ntimes-1;time>=0;time--)
      
    }  // if(comp_grad==1)

  }  //  if(index<nnzp)
  
}




















