




#include <hip/hip_runtime.h>
#include <cmath>

#include <stdio.h>


#define TWO_PI_GAMMA (2.675128976384781e+08)



__device__ void multiply_2spinors(double a1_re,double a1_im,double b1_re,double b1_im,
                                  double a2_re,double a2_im,double b2_re,double b2_im,
                                  double *a3_re,double *a3_im,double *b3_re,double *b3_im)
{
  *a3_re = a2_re*a1_re - a2_im*a1_im - b2_re*b1_re - b2_im*b1_im;
  *a3_im = a2_im*a1_re + a2_re*a1_im + b2_im*b1_re - b2_re*b1_im;
  
  *b3_re = b2_re*a1_re - b2_im*a1_im + a2_re*b1_re + a2_im*b1_im;
  *b3_im = b2_im*a1_re + b2_re*a1_im - a2_im*b1_re + a2_re*b1_im;
}




__device__ void multiply_3spinors(double a1_re,double a1_im,double b1_re,double b1_im,
                                  double a2_re,double a2_im,double b2_re,double b2_im,
                                  double a3_re,double a3_im,double b3_re,double b3_im,
                                  double *a4_re,double *a4_im,double *b4_re,double *b4_im)
{
  
  double tmp1_re = a2_re*a1_re - a2_im*a1_im - b2_re*b1_re - b2_im*b1_im;
  double tmp1_im = a2_im*a1_re + a2_re*a1_im + b2_im*b1_re - b2_re*b1_im;
  
  double tmp2_re = b2_re*a1_re - b2_im*a1_im + a2_re*b1_re + a2_im*b1_im;
  double tmp2_im = b2_im*a1_re + b2_re*a1_im - a2_im*b1_re + a2_re*b1_im;
  
  *a4_re = a3_re*tmp1_re - a3_im*tmp1_im - b3_re*tmp2_re - b3_im*tmp2_im;
  *a4_im = a3_im*tmp1_re + a3_re*tmp1_im + b3_im*tmp2_re - b3_re*tmp2_im;
  
  *b4_re = b3_re*tmp1_re - b3_im*tmp1_im + a3_re*tmp2_re + a3_im*tmp2_im;
  *b4_im = b3_im*tmp1_re + b3_re*tmp1_im - a3_im*tmp2_re + a3_re*tmp2_im;    
}



// set inc_gblip==1 for incorporation of the gradient blip in the CK parameters
__device__ void compute_caley_klein_params(double b1_re,double b1_im,double deltat,
					   double gblip_re,double gblip_im,int inc_gblip,
					   double *b1_mag,double *b1_pha,double *cos_b1_pha,double *sin_b1_pha,
					   double *phi,double *cosphi,double *sinphi,
					   double *a_re,double *a_im,double *b_re,double *b_im)
{

  *b1_mag=sqrt( b1_re*b1_re + b1_im*b1_im );
    
  *phi=-deltat*(*b1_mag)*TWO_PI_GAMMA;  // see Pauly et al.
  sincos( (*phi)/2.0,sinphi,cosphi );
  
  if(inc_gblip==1){
    *a_re = (*cosphi) * gblip_re;  // a2 = a2 * Qblip
    *a_im = (*cosphi) * gblip_im;
  }else{
    *a_re = (*cosphi);
    *a_im = 0.0;
  }


  *b1_pha = atan2( b1_im,b1_re );
  sincos( *b1_pha,sin_b1_pha,cos_b1_pha );
    
  double tmp_re= (*sin_b1_pha)*(*sinphi);
  double tmp_im=-(*cos_b1_pha)*(*sinphi);
  
  if(inc_gblip==1){
    *b_re = tmp_re*gblip_re + tmp_im*gblip_im;  // b2 = b2 * conj(Qblip)
    *b_im = tmp_im*gblip_re - tmp_re*gblip_im;
  }else{
    *b_re = tmp_re;
    *b_im = tmp_im;
  }

}




__device__  void compute_jacobian_Q_matrix( double b1maps_re,double b1maps_im,
					    double cos_btot_pha,double sin_btot_pha,
					    double cosphi,double sinphi,
					    double btot_mag,double deltat,double sumsinc,
					    double gblip_re,double gblip_im,
					    double *da_dre_re,double *da_dre_im,double *da_dim_re,double *da_dim_im,
					    double *db_dre_re,double *db_dre_im,double *db_dim_re,double *db_dim_im )
{

  double tmp_re = b1maps_re*cos_btot_pha + b1maps_im*sin_btot_pha;
  double tmp_im = b1maps_im*cos_btot_pha - b1maps_re*sin_btot_pha;
  
  double dphi_dre = -deltat*abs(sumsinc)*TWO_PI_GAMMA*tmp_re;
  double dphi_dim =  deltat*abs(sumsinc)*TWO_PI_GAMMA*tmp_im;

  // derivative of alpha
  double tmp_re_re = -0.5*sinphi*dphi_dre;
  double tmp_re_im = 0.0;

  double tmp_im_re = -0.5*sinphi*dphi_dim;
  double tmp_im_im = 0.0;
  
  // gradient blip: alpha = alpha * gblip
  *da_dre_re = tmp_re_re * gblip_re;
  *da_dre_im = tmp_re_re * gblip_im;

  *da_dim_re = tmp_im_re * gblip_re;
  *da_dim_im = tmp_im_re * gblip_im;
    
  // derivative of the term exp(1j*angle(btot))
  double dexppha_dre_re=0.0,dexppha_dre_im=0.0,dexppha_dim_re=0.0,dexppha_dim_im=0.0;
  if(btot_mag>0){
    dexppha_dre_re = sumsinc/btot_mag*( b1maps_re - cos_btot_pha*tmp_re );
    dexppha_dre_im = sumsinc/btot_mag*( b1maps_im - sin_btot_pha*tmp_re );

    dexppha_dim_re = sumsinc/btot_mag*( -b1maps_im + cos_btot_pha*tmp_im );
    dexppha_dim_im = sumsinc/btot_mag*(  b1maps_re + sin_btot_pha*tmp_im );
  }

  // derivative of beta
  tmp_re_re = sinphi*dexppha_dre_im + 0.5*cosphi*sin_btot_pha*dphi_dre;
  tmp_re_im = -sinphi*dexppha_dre_re - 0.5*cosphi*cos_btot_pha*dphi_dre;

  tmp_im_re = sinphi*dexppha_dim_im + 0.5*cosphi*sin_btot_pha*dphi_dim;
  tmp_im_im = -sinphi*dexppha_dim_re - 0.5*cosphi*cos_btot_pha*dphi_dim;

  // gradient blip: beta = beta * conj(gblip)
  *db_dre_re = tmp_re_re*gblip_re + tmp_re_im*gblip_im;
  *db_dre_im = tmp_re_im*gblip_re - tmp_re_re*gblip_im;

  *db_dim_re = tmp_im_re*gblip_re + tmp_im_im*gblip_im;
  *db_dim_im = tmp_im_im*gblip_re - tmp_im_re*gblip_im;
  
}





__device__ void compute_hessian_Q_matrix( double cosphi,double sinphi,
					  double b1maps_j_re,double b1maps_j_im,
					  double b1maps_k_re,double b1maps_k_im,
					  double cos_btot_pha,double sin_btot_pha, double btot_mag,
					  double deltat,double sumsinc,				
					  double gblip_re,double gblip_im,
					  double *h_a_1_re,double *h_a_1_im,double *h_b_1_re,double *h_b_1_im,
					  double *h_a_2_re,double *h_a_2_im,double *h_b_2_re,double *h_b_2_im,
					  double *h_a_3_re,double *h_a_3_im,double *h_b_3_re,double *h_b_3_im )
{

  double tmp_j_re = b1maps_j_re*cos_btot_pha + b1maps_j_im*sin_btot_pha;
  double tmp_j_im = b1maps_j_im*cos_btot_pha - b1maps_j_re*sin_btot_pha;

  double tmp_k_re = b1maps_k_re*cos_btot_pha + b1maps_k_im*sin_btot_pha;
  double tmp_k_im = b1maps_k_im*cos_btot_pha - b1maps_k_re*sin_btot_pha;

  double dphi_dre_j = -deltat*abs(sumsinc)*TWO_PI_GAMMA*tmp_j_re;
  double dphi_dim_j =  deltat*abs(sumsinc)*TWO_PI_GAMMA*tmp_j_im;

  double dphi_dre_k = -deltat*abs(sumsinc)*TWO_PI_GAMMA*tmp_k_re;
  double dphi_dim_k =  deltat*abs(sumsinc)*TWO_PI_GAMMA*tmp_k_im;


  // HESSIAN OF PHI
  double norm=0.0;
  if(btot_mag>0.0)
    norm=-deltat*sumsinc*sumsinc/btot_mag*TWO_PI_GAMMA;

  // b1maps(:,j).*conj(b1maps(:,k))
  double tmp_re = b1maps_j_re*b1maps_k_re + b1maps_j_im*b1maps_k_im;
  double tmp_im = b1maps_j_im*b1maps_k_re - b1maps_j_re*b1maps_k_im;

  double h_phi_1 = norm*( tmp_re - tmp_j_re*tmp_k_re );  
  double h_phi_3 = norm*( tmp_re - tmp_j_im*tmp_k_im );
  double h_phi_2 = norm*( -tmp_im + tmp_j_im*tmp_k_re );


  // HESSIAN OF ALPHA
  double tmp_h_1_re = -0.25*cosphi*dphi_dre_j*dphi_dre_k - 0.5*sinphi*h_phi_1;  // hessian term #1 (dre_dre)
  double tmp_h_3_re = -0.25*cosphi*dphi_dim_j*dphi_dim_k - 0.5*sinphi*h_phi_3;  // hessian term #3 (dim_dim) 
  double tmp_h_2_re = -0.25*cosphi*dphi_dim_j*dphi_dre_k - 0.5*sinphi*h_phi_2;  // hessian term #2 (dre_dim)

  double tmp_h_1_im = 0.0;  // alpha is real when B0=0
  double tmp_h_2_im = 0.0;
  double tmp_h_3_im = 0.0;

  *h_a_1_re = tmp_h_1_re * gblip_re;
  *h_a_1_im = tmp_h_1_re * gblip_im;

  *h_a_2_re = tmp_h_2_re * gblip_re;
  *h_a_2_im = tmp_h_2_re * gblip_im;

  *h_a_3_re = tmp_h_3_re * gblip_re;
  *h_a_3_im = tmp_h_3_re * gblip_im;
  

  // HESSIAN OF THE TERM exp(1j*angle(btot))
  double V=btot_mag;
  double V_re=V;
  double V_im=0.0;

  double h_exppha_1_re=0.0,h_exppha_1_im=0.0;
  double h_exppha_2_re=0.0,h_exppha_2_im=0.0;
  double h_exppha_3_re=0.0,h_exppha_3_im=0.0;

  double dexppha_dre_j_re=0.0, dexppha_dre_j_im=0.0;
  double dexppha_dim_j_re=0.0, dexppha_dim_j_im=0.0;

  double dexppha_dre_k_re=0.0, dexppha_dre_k_im=0.0;
  double dexppha_dim_k_re=0.0, dexppha_dim_k_im=0.0;

  if(V>0){

    dexppha_dre_j_re = sumsinc/btot_mag*( b1maps_j_re - cos_btot_pha*tmp_j_re );
    dexppha_dre_j_im = sumsinc/btot_mag*( b1maps_j_im - sin_btot_pha*tmp_j_re );  
    dexppha_dim_j_re = sumsinc/btot_mag*( -b1maps_j_im + cos_btot_pha*tmp_j_im );
    dexppha_dim_j_im = sumsinc/btot_mag*(  b1maps_j_re + sin_btot_pha*tmp_j_im );

    dexppha_dre_k_re = sumsinc/btot_mag*( b1maps_k_re - cos_btot_pha*tmp_k_re );
    dexppha_dre_k_im = sumsinc/btot_mag*( b1maps_k_im - sin_btot_pha*tmp_k_re );  
    dexppha_dim_k_re = sumsinc/btot_mag*( -b1maps_k_im + cos_btot_pha*tmp_k_im );
    dexppha_dim_k_im = sumsinc/btot_mag*(  b1maps_k_re + sin_btot_pha*tmp_k_im );

    // term #1
    double U_re = sumsinc*( b1maps_j_re - cos_btot_pha*tmp_j_re );
    double U_im = sumsinc*( b1maps_j_im - sin_btot_pha*tmp_j_re );

    double dU_re = ( -dexppha_dre_k_re*tmp_j_re - cos_btot_pha*(b1maps_j_re*dexppha_dre_k_re + b1maps_j_im*dexppha_dre_k_im) )*sumsinc;
    double dU_im = ( -dexppha_dre_k_im*tmp_j_re - sin_btot_pha*(b1maps_j_re*dexppha_dre_k_re + b1maps_j_im*dexppha_dre_k_im) )*sumsinc;

    double dV_re = sumsinc*tmp_k_re;
    double dV_im = 0.0;
    
    h_exppha_1_re = ( dU_re*V_re - dU_im*V_im - dV_re*U_re + dV_im*U_im )/(V*V);
    h_exppha_1_im = ( dU_im*V_re + dU_re*V_im - dV_im*U_re - dV_re*U_im )/(V*V);
    
    // term #3
    U_re = sumsinc*( -b1maps_j_im + cos_btot_pha*tmp_j_im );
    U_im = sumsinc*(  b1maps_j_re + sin_btot_pha*tmp_j_im );

    dU_re = sumsinc*( dexppha_dim_k_re*tmp_j_im + cos_btot_pha*(b1maps_j_im*dexppha_dim_k_re - b1maps_j_re*dexppha_dim_k_im) );
    dU_im = sumsinc*( dexppha_dim_k_im*tmp_j_im + sin_btot_pha*(b1maps_j_im*dexppha_dim_k_re - b1maps_j_re*dexppha_dim_k_im) );

    dV_re = -sumsinc*tmp_k_im;
    dV_im = 0.0;

    h_exppha_3_re = ( dU_re*V_re - dU_im*V_im - dV_re*U_re + dV_im*U_im )/(V*V);
    h_exppha_3_im = ( dU_im*V_re + dU_re*V_im - dV_im*U_re - dV_re*U_im )/(V*V);
    
    // term #2
    U_re = sumsinc*( -b1maps_j_im + cos_btot_pha*tmp_j_im );
    U_im = sumsinc*(  b1maps_j_re + sin_btot_pha*tmp_j_im );

    dU_re = sumsinc*( dexppha_dre_k_re*tmp_j_im + cos_btot_pha*(b1maps_j_im*dexppha_dre_k_re - b1maps_j_re*dexppha_dre_k_im) );
    dU_im = sumsinc*( dexppha_dre_k_im*tmp_j_im + sin_btot_pha*(b1maps_j_im*dexppha_dre_k_re - b1maps_j_re*dexppha_dre_k_im) );

    dV_re = sumsinc*tmp_k_re;
    dV_im = 0.0;

    h_exppha_2_re = ( dU_re*V_re - dU_im*V_im - dV_re*U_re + dV_im*U_im )/(V*V);
    h_exppha_2_im = ( dU_im*V_re + dU_re*V_im - dV_im*U_re - dV_re*U_im )/(V*V);
  }


  // HESSIAN OF BETA

  // term #1
  double tmp1_re =  0.5*cosphi*( dphi_dre_k*dexppha_dre_j_im + dphi_dre_j*dexppha_dre_k_im );
  double tmp1_im = -0.5*cosphi*( dphi_dre_k*dexppha_dre_j_re + dphi_dre_j*dexppha_dre_k_re );
                                         
  double tmp2_re =  sinphi*h_exppha_1_im;
  double tmp2_im = -sinphi*h_exppha_1_re;

  double tmp3_re = -0.25*sinphi*dphi_dre_k*dphi_dre_j*sin_btot_pha;
  double tmp3_im =  0.25*sinphi*dphi_dre_k*dphi_dre_j*cos_btot_pha;

  double tmp4_re =  0.5*cosphi*h_phi_1*sin_btot_pha;
  double tmp4_im = -0.5*cosphi*h_phi_1*cos_btot_pha;

  tmp_h_1_re=tmp1_re + tmp2_re + tmp3_re + tmp4_re;
  tmp_h_1_im=tmp1_im + tmp2_im + tmp3_im + tmp4_im;
  
  // term #3
  tmp1_re =  0.5*cosphi*( dphi_dim_k*dexppha_dim_j_im + dphi_dim_j*dexppha_dim_k_im );
  tmp1_im = -0.5*cosphi*( dphi_dim_k*dexppha_dim_j_re + dphi_dim_j*dexppha_dim_k_re );

  tmp2_re =  sinphi*h_exppha_3_im;
  tmp2_im = -sinphi*h_exppha_3_re;                                         

  tmp3_re = -0.25*sinphi*dphi_dim_k*dphi_dim_j*sin_btot_pha;
  tmp3_im =  0.25*sinphi*dphi_dim_k*dphi_dim_j*cos_btot_pha;

  tmp4_re =  0.5*cosphi*h_phi_3*sin_btot_pha;
  tmp4_im = -0.5*cosphi*h_phi_3*cos_btot_pha;

  tmp_h_3_re=tmp1_re + tmp2_re + tmp3_re + tmp4_re;
  tmp_h_3_im=tmp1_im + tmp2_im + tmp3_im + tmp4_im;
  
  // term #2
  tmp1_re =  0.5*cosphi*( dphi_dre_k*dexppha_dim_j_im + dphi_dim_j*dexppha_dre_k_im );
  tmp1_im = -0.5*cosphi*( dphi_dre_k*dexppha_dim_j_re + dphi_dim_j*dexppha_dre_k_re );

  tmp2_re =  sinphi*h_exppha_2_im;
  tmp2_im = -sinphi*h_exppha_2_re;                          

  tmp3_re = -0.25*sinphi*dphi_dre_k*dphi_dim_j*sin_btot_pha;
  tmp3_im =  0.25*sinphi*dphi_dre_k*dphi_dim_j*cos_btot_pha;               

  tmp4_re =  0.5*cosphi*h_phi_2*sin_btot_pha;
  tmp4_im = -0.5*cosphi*h_phi_2*cos_btot_pha;

  tmp_h_2_re=tmp1_re + tmp2_re + tmp3_re + tmp4_re;
  tmp_h_2_im=tmp1_im + tmp2_im + tmp3_im + tmp4_im;


  // incorporate gradient blip into the hessian of b
  *h_b_1_re = tmp_h_1_re*gblip_re + tmp_h_1_im*gblip_im;
  *h_b_1_im = tmp_h_1_im*gblip_re - tmp_h_1_re*gblip_im;

  *h_b_2_re = tmp_h_2_re*gblip_re + tmp_h_2_im*gblip_im;
  *h_b_2_im = tmp_h_2_im*gblip_re - tmp_h_2_re*gblip_im;

  *h_b_3_re = tmp_h_3_re*gblip_re + tmp_h_3_im*gblip_im;
  *h_b_3_im = tmp_h_3_im*gblip_re - tmp_h_3_re*gblip_im;  

}





__global__ void hessian_LFA_noB0_cudaKernel( double *a_re,double *a_im,double *b_re,double *b_im,
					     double *ha_re,double *ha_im,
					     double *hb_re,double *hb_im,
					     double *ja_re_re,double *ja_re_im,double *ja_im_re,double *ja_im_im,
					     double *jb_re_re,double *jb_re_im,double *jb_im_re,double *jb_im_im,					     
					     double *a_forw_re,double *a_forw_im,double *b_forw_re,double *b_forw_im,
					     double *btotspokes_re,double *btotspokes_im,
					     double *q_gblips_re,double *q_gblips_im,
					     double *b1maps_re,double *b1maps_im,
					     int nnzp,int nspokes,int ncoils,double sumsinc,double deltat )
{


  int index=threadIdx.x + blockIdx.x * blockDim.x;  //  global thread index == voxel index  


  if(index<nnzp){

    double atot_re=1.0;
    double atot_im=0.0;
    
    double btot_re=0.0;
    double btot_im=0.0;

    a_forw_re[ index ] = 1.0;
    a_forw_im[ index ] = 0.0;
    
    b_forw_re[ index ] = 0.0;
    b_forw_im[ index ] = 0.0;
    
     
    int ncunk=nspokes*ncoils;

    double a_1_re,a_1_im,b_1_re,b_1_im,b1_1_mag,b1_1_pha,cos_b1_1_pha,sin_b1_1_pha,phi_1,cos_phi_1,sin_phi_1;
    
    // FORWARD BLOCH SIMULATION    
    for(int sp=0;sp<nspokes;sp++){
      
      // CK parameters
      compute_caley_klein_params( btotspokes_re[index+sp*nnzp],btotspokes_im[index+sp*nnzp],deltat,
				  q_gblips_re[index+sp*nnzp],q_gblips_im[index+sp*nnzp],1,
				  &b1_1_mag,&b1_1_pha,&cos_b1_1_pha,&sin_b1_1_pha,
				  &phi_1,&cos_phi_1,&sin_phi_1,
				  &a_1_re,&a_1_im,&b_1_re,&b_1_im );
      
      // forward Q products (one step late)
      if(sp<nspokes-1){

	multiply_2spinors( atot_re ,atot_im ,btot_re ,btot_im  ,
			   a_1_re  ,a_1_im  ,b_1_re  ,b_1_im   ,
			   &atot_re,&atot_im,&btot_re,&btot_im );

	a_forw_re[ index+(sp+1)*nnzp ] = atot_re;
	a_forw_im[ index+(sp+1)*nnzp ] = atot_im;

	b_forw_re[ index+(sp+1)*nnzp ] = btot_re;
	b_forw_im[ index+(sp+1)*nnzp ] = btot_im;
	
      }
                  
    }  // for(int sp=0;sp<nspokes;sp++)


    // BACKWARD BLOCH SIMULATION         
    atot_re=1.0;
    atot_im=0.0;
    
    btot_re=0.0;
    btot_im=0.0;
   
    double a_2_re,a_2_im,b_2_re,b_2_im,b1_2_mag,b1_2_pha,cos_b1_2_pha,sin_b1_2_pha,phi_2,cos_phi_2,sin_phi_2;
    double a_3_re,a_3_im,b_3_re,b_3_im,b1_3_mag,b1_3_pha,cos_b1_3_pha,sin_b1_3_pha,phi_3,cos_phi_3,sin_phi_3;

    double da_1_dre_re,da_1_dre_im,da_1_dim_re,da_1_dim_im;
    double db_1_dre_re,db_1_dre_im,db_1_dim_re,db_1_dim_im;

    double A_TMP1_RE_re[8],A_TMP1_RE_im[8];
    double B_TMP1_RE_re[8],B_TMP1_RE_im[8];
    
    double A_TMP1_IM_re[8],A_TMP1_IM_im[8];
    double B_TMP1_IM_re[8],B_TMP1_IM_im[8];

    double A_TMP2_RE_re[8],A_TMP2_RE_im[8];
    double B_TMP2_RE_re[8],B_TMP2_RE_im[8];
    
    double A_TMP2_IM_re[8],A_TMP2_IM_im[8];
    double B_TMP2_IM_re[8],B_TMP2_IM_im[8];

    for(int sp=nspokes-1;sp>=0;sp--){

      compute_caley_klein_params( btotspokes_re[index+sp*nnzp],btotspokes_im[index+sp*nnzp],deltat,
				  q_gblips_re[index+sp*nnzp],q_gblips_im[index+sp*nnzp],1,
				  &b1_1_mag,&b1_1_pha,&cos_b1_1_pha,&sin_b1_1_pha,
				  &phi_1,&cos_phi_1,&sin_phi_1,
				  &a_1_re,&a_1_im,&b_1_re,&b_1_im );
            
      int ind=index + sp*nnzp;
      
      for(int j=0;j<ncoils;j++){
	
	int ind_j=j + sp*ncoils;
	int ind2=index + (j+sp*ncoils)*nnzp;

	// Jacobian (sandwitch product of backward Q product, Jacobian of current Q matrix and forward Q products) 
	compute_jacobian_Q_matrix( b1maps_re[index+j*nnzp],b1maps_im[index+j*nnzp],
				   cos_b1_1_pha,sin_b1_1_pha,
				   cos_phi_1,sin_phi_1,
				   b1_1_mag,deltat,sumsinc,
				   q_gblips_re[index+sp*nnzp],q_gblips_im[index+sp*nnzp],
				   &da_1_dre_re,&da_1_dre_im,&da_1_dim_re,&da_1_dim_im ,
				   &db_1_dre_re,&db_1_dre_im,&db_1_dim_re,&db_1_dim_im );

	
	multiply_3spinors( a_forw_re[ind] ,a_forw_im[ind] ,b_forw_re[ind] ,b_forw_im[ind] ,
			   da_1_dre_re    ,da_1_dre_im    ,db_1_dre_re    ,db_1_dre_im    ,
			   atot_re        ,atot_im        ,btot_re        ,btot_im        ,
			   &ja_re_re[ind2],&ja_re_im[ind2],&jb_re_re[ind2],&jb_re_im[ind2]);

	multiply_3spinors( a_forw_re[ind] ,a_forw_im[ind] ,b_forw_re[ind] ,b_forw_im[ind] ,
			   da_1_dim_re    ,da_1_dim_im    ,db_1_dim_re    ,db_1_dim_im    ,
			   atot_re        ,atot_im        ,btot_re        ,btot_im        ,
			   &ja_im_re[ind2],&ja_im_im[ind2],&jb_im_re[ind2],&jb_im_im[ind2]);


	// Hessian term (within a single spoke)
	for(int k=0;k<ncoils;k++){
	  int ind_k=k + sp*ncoils;

	  double h_a_1_re,h_a_1_im,h_b_1_re,h_b_1_im;
	  double h_a_2_re,h_a_2_im,h_b_2_re,h_b_2_im;
	  double h_a_3_re,h_a_3_im,h_b_3_re,h_b_3_im;

	  compute_hessian_Q_matrix(cos_phi_1,sin_phi_1,
				   b1maps_re[index+j*nnzp],b1maps_im[index+j*nnzp],
				   b1maps_re[index+k*nnzp],b1maps_im[index+k*nnzp],
				   cos_b1_1_pha,sin_b1_1_pha,b1_1_mag,
				   deltat,sumsinc,
				   q_gblips_re[index+sp*nnzp],q_gblips_im[index+sp*nnzp],
				   &h_a_1_re,&h_a_1_im,&h_b_1_re,&h_b_1_im,
				   &h_a_2_re,&h_a_2_im,&h_b_2_re,&h_b_2_im,
				   &h_a_3_re,&h_a_3_im,&h_b_3_re,&h_b_3_im );

	  
	  double tmp_a_1_re,tmp_a_1_im,tmp_b_1_re,tmp_b_1_im;
	  double tmp_a_2_re,tmp_a_2_im,tmp_b_2_re,tmp_b_2_im;
	  double tmp_a_3_re,tmp_a_3_im,tmp_b_3_re,tmp_b_3_im;

	  multiply_3spinors( a_forw_re[ind],a_forw_im[ind],b_forw_re[ind],b_forw_im[ind],
			     h_a_1_re      ,h_a_1_im      ,h_b_1_re      ,h_b_1_im      ,
			     atot_re       ,atot_im       ,btot_re       ,btot_im       ,
			     &tmp_a_1_re   ,&tmp_a_1_im   ,&tmp_b_1_re   ,&tmp_b_1_im   );

	  multiply_3spinors( a_forw_re[ind],a_forw_im[ind],b_forw_re[ind],b_forw_im[ind],
			     h_a_2_re      ,h_a_2_im      ,h_b_2_re      ,h_b_2_im      ,
			     atot_re       ,atot_im       ,btot_re       ,btot_im       ,
			     &tmp_a_2_re   ,&tmp_a_2_im   ,&tmp_b_2_re   ,&tmp_b_2_im   );

	  multiply_3spinors( a_forw_re[ind],a_forw_im[ind],b_forw_re[ind],b_forw_im[ind],
			     h_a_3_re      ,h_a_3_im      ,h_b_3_re      ,h_b_3_im      ,
			     atot_re       ,atot_im       ,btot_re       ,btot_im       ,
			     &tmp_a_3_re   ,&tmp_a_3_im   ,&tmp_b_3_re   ,&tmp_b_3_im   );
          
	  // fill ha
	  ha_re[index + (ind_k       + ind_j        *2*ncunk)*nnzp] = tmp_a_1_re;
	  ha_re[index + (ind_j+ncunk + ind_k        *2*ncunk)*nnzp] = tmp_a_2_re;
	  ha_re[index + (ind_k       + (ind_j+ncunk)*2*ncunk)*nnzp] = tmp_a_2_re;
	  ha_re[index + (ind_k+ncunk + (ind_j+ncunk)*2*ncunk)*nnzp] = tmp_a_3_re;                    

	  ha_im[index + (ind_k       + ind_j        *2*ncunk)*nnzp] = tmp_a_1_im;
	  ha_im[index + (ind_j+ncunk + ind_k        *2*ncunk)*nnzp] = tmp_a_2_im;
	  ha_im[index + (ind_k       + (ind_j+ncunk)*2*ncunk)*nnzp] = tmp_a_2_im;
	  ha_im[index + (ind_k+ncunk + (ind_j+ncunk)*2*ncunk)*nnzp] = tmp_a_3_im;                    
	   
	  // fill hb
	  hb_re[index + (ind_k       + ind_j        *2*ncunk)*nnzp] = tmp_b_1_re;
	  hb_re[index + (ind_j+ncunk + ind_k        *2*ncunk)*nnzp] = tmp_b_2_re;
	  hb_re[index + (ind_k       + (ind_j+ncunk)*2*ncunk)*nnzp] = tmp_b_2_re;
	  hb_re[index + (ind_k+ncunk + (ind_j+ncunk)*2*ncunk)*nnzp] = tmp_b_3_re;

	  hb_im[index + (ind_k       + ind_j        *2*ncunk)*nnzp] = tmp_b_1_im;
	  hb_im[index + (ind_j+ncunk + ind_k        *2*ncunk)*nnzp] = tmp_b_2_im;
	  hb_im[index + (ind_k       + (ind_j+ncunk)*2*ncunk)*nnzp] = tmp_b_2_im;
	  hb_im[index + (ind_k+ncunk + (ind_j+ncunk)*2*ncunk)*nnzp] = tmp_b_3_im;

	}
        
	
	// Jacobian terms (derivatives taken for different spokes)
	if(nspokes>0){
	  
	  // product between left-hand jacobian term and backward Q products
	  multiply_2spinors( da_1_dre_re     ,da_1_dre_im     ,db_1_dre_re     ,db_1_dre_im      ,
			     atot_re         ,atot_im         ,btot_re         ,btot_im          ,
			     &A_TMP1_RE_re[j],&A_TMP1_RE_im[j],&B_TMP1_RE_re[j],&B_TMP1_RE_im[j] );
	  
	  multiply_2spinors( da_1_dim_re     ,da_1_dim_im     ,db_1_dim_re     ,db_1_dim_im      ,
			     atot_re         ,atot_im         ,btot_re         ,btot_im          ,
			     &A_TMP1_IM_re[j],&A_TMP1_IM_im[j],&B_TMP1_IM_re[j],&B_TMP1_IM_im[j] );	  	 

	}

      }  // if(int j=0;j<ncoils;j++)


      if(nspokes>0){
        
	for(int sp2=sp-1;sp2>=0;sp2--){
	  
	  // compute the product of Q matrices sandwitched between indices "sp" and "sp2"
	  double a_sandw_re=1.0;
	  double a_sandw_im=0.0;

	  double b_sandw_re=0.0;
	  double b_sandw_im=0.0;

	  for(int sp3=sp-1;sp3>=sp2+1;sp3--){

	    compute_caley_klein_params( btotspokes_re[index+sp3*nnzp],btotspokes_im[index+sp3*nnzp],deltat,
					q_gblips_re[index+sp3*nnzp],q_gblips_im[index+sp3*nnzp],1,
					&b1_3_mag,&b1_3_pha,&cos_b1_3_pha,&sin_b1_3_pha,
					&phi_3,&cos_phi_3,&sin_phi_3,
					&a_3_re,&a_3_im,&b_3_re,&b_3_im );

	    multiply_2spinors(a_3_re     ,a_3_im     ,b_3_re     ,b_3_im      ,
			      a_sandw_re ,a_sandw_im ,b_sandw_re ,b_sandw_im  ,
			      &a_sandw_re,&a_sandw_im,&b_sandw_re,&b_sandw_im );
	  }			      
	  

	  // product between sandwitch term, right-hand jacobian term and forward Q products 
	  for(int j=0;j<ncoils;j++){
	    ind=index + (j+sp2*ncoils)*nnzp;

	    compute_caley_klein_params( btotspokes_re[index+sp2*nnzp],btotspokes_im[index+sp2*nnzp],deltat,
					q_gblips_re[index+sp2*nnzp],q_gblips_im[index+sp2*nnzp],1,
					&b1_2_mag,&b1_2_pha,&cos_b1_2_pha,&sin_b1_2_pha,
					&phi_2,&cos_phi_2,&sin_phi_2,
					&a_2_re,&a_2_im,&b_2_re,&b_2_im );

	    double da_2_dre_re,da_2_dre_im,da_2_dim_re,da_2_dim_im;
	    double db_2_dre_re,db_2_dre_im,db_2_dim_re,db_2_dim_im;

	    compute_jacobian_Q_matrix( b1maps_re[index+j*nnzp],b1maps_im[index+j*nnzp],
				       cos_b1_2_pha,sin_b1_2_pha,
				       cos_phi_2,sin_phi_2,
				       b1_2_mag,deltat,sumsinc,
				       q_gblips_re[index+sp2*nnzp],q_gblips_im[index+sp2*nnzp],
				       &da_2_dre_re,&da_2_dre_im,&da_2_dim_re,&da_2_dim_im ,
				       &db_2_dre_re,&db_2_dre_im,&db_2_dim_re,&db_2_dim_im );

	    multiply_3spinors( a_forw_re[index+sp2*nnzp],a_forw_im[index+sp2*nnzp],b_forw_re[index+sp2*nnzp],b_forw_im[index+sp2*nnzp],
			       da_2_dre_re              ,da_2_dre_im              ,db_2_dre_re              ,db_2_dre_im              ,
			       a_sandw_re               ,a_sandw_im               ,b_sandw_re               ,b_sandw_im               ,
			       &A_TMP2_RE_re[j]         ,&A_TMP2_RE_im[j]         ,&B_TMP2_RE_re[j]         ,&B_TMP2_RE_im[j]         );

	    multiply_3spinors( a_forw_re[index+sp2*nnzp],a_forw_im[index+sp2*nnzp],b_forw_re[index+sp2*nnzp],b_forw_im[index+sp2*nnzp],
			       da_2_dim_re              ,da_2_dim_im              ,db_2_dim_re              ,db_2_dim_im              ,
			       a_sandw_re               ,a_sandw_im               ,b_sandw_re               ,b_sandw_im               ,
			       &A_TMP2_IM_re[j]         ,&A_TMP2_IM_im[j]         ,&B_TMP2_IM_re[j]         ,&B_TMP2_IM_im[j]         );
	  }
	  
	  // outter products between the columns of the TMP1 and TMP2 jacobian terms
	  double tmp_a_1_re,tmp_a_1_im,tmp_b_1_re,tmp_b_1_im;
	  double tmp_a_2_re,tmp_a_2_im,tmp_b_2_re,tmp_b_2_im;
	  double tmp_a_3_re,tmp_a_3_im,tmp_b_3_re,tmp_b_3_im;
	  double tmp_a_4_re,tmp_a_4_im,tmp_b_4_re,tmp_b_4_im;

	  for(int j=0;j<ncoils;j++){
	  
	    for(int k=0;k<ncoils;k++){
             	      
	      multiply_2spinors( A_TMP2_RE_re[k],A_TMP2_RE_im[k],B_TMP2_RE_re[k],B_TMP2_RE_im[k] ,
				 A_TMP1_RE_re[j],A_TMP1_RE_im[j],B_TMP1_RE_re[j],B_TMP1_RE_im[j] ,
				 &tmp_a_1_re    ,&tmp_a_1_im    ,&tmp_b_1_re    ,&tmp_b_1_im     );

	      multiply_2spinors( A_TMP2_RE_re[k],A_TMP2_RE_im[k],B_TMP2_RE_re[k],B_TMP2_RE_im[k] ,
				 A_TMP1_IM_re[j],A_TMP1_IM_im[j],B_TMP1_IM_re[j],B_TMP1_IM_im[j] ,
				 &tmp_a_2_re    ,&tmp_a_2_im    ,&tmp_b_2_re    ,&tmp_b_2_im     );

	      multiply_2spinors( A_TMP2_IM_re[k],A_TMP2_IM_im[k],B_TMP2_IM_re[k],B_TMP2_IM_im[k] ,
				 A_TMP1_RE_re[j],A_TMP1_RE_im[j],B_TMP1_RE_re[j],B_TMP1_RE_im[j] ,
				 &tmp_a_3_re    ,&tmp_a_3_im    ,&tmp_b_3_re    ,&tmp_b_3_im     );

	      multiply_2spinors( A_TMP2_IM_re[k],A_TMP2_IM_im[k],B_TMP2_IM_re[k],B_TMP2_IM_im[k] ,
				 A_TMP1_IM_re[j],A_TMP1_IM_im[j],B_TMP1_IM_re[j],B_TMP1_IM_im[j] ,
				 &tmp_a_4_re    ,&tmp_a_4_im    ,&tmp_b_4_re    ,&tmp_b_4_im     );


	      // half of the Hessian terms are filled by symmetry
	      int ind_j2 = j + sp*ncoils;
	      int ind_k3 = k + sp2*ncoils;

	      ha_re[index + (ind_k3       + ind_j2        *2*ncunk)*nnzp] = ha_re[index + (ind_j2       + ind_k3        *2*ncunk)*nnzp] = tmp_a_1_re;
	      ha_re[index + (ind_k3       + (ind_j2+ncunk)*2*ncunk)*nnzp] = ha_re[index + (ind_j2+ncunk + ind_k3        *2*ncunk)*nnzp] = tmp_a_2_re;
	      ha_re[index + (ind_k3+ncunk + ind_j2        *2*ncunk)*nnzp] = ha_re[index + (ind_j2       + (ind_k3+ncunk)*2*ncunk)*nnzp] = tmp_a_3_re;
	      ha_re[index + (ind_k3+ncunk + (ind_j2+ncunk)*2*ncunk)*nnzp] = ha_re[index + (ind_j2+ncunk + (ind_k3+ncunk)*2*ncunk)*nnzp] = tmp_a_4_re;

	      ha_im[index + (ind_k3       + ind_j2        *2*ncunk)*nnzp] = ha_im[index + (ind_j2       + ind_k3        *2*ncunk)*nnzp] = tmp_a_1_im;
	      ha_im[index + (ind_k3       + (ind_j2+ncunk)*2*ncunk)*nnzp] = ha_im[index + (ind_j2+ncunk + ind_k3        *2*ncunk)*nnzp] = tmp_a_2_im;
	      ha_im[index + (ind_k3+ncunk + ind_j2        *2*ncunk)*nnzp] = ha_im[index + (ind_j2       + (ind_k3+ncunk)*2*ncunk)*nnzp] = tmp_a_3_im;
	      ha_im[index + (ind_k3+ncunk + (ind_j2+ncunk)*2*ncunk)*nnzp] = ha_im[index + (ind_j2+ncunk + (ind_k3+ncunk)*2*ncunk)*nnzp] = tmp_a_4_im;


	      hb_re[index + (ind_k3       + ind_j2        *2*ncunk)*nnzp] = hb_re[index + (ind_j2       + ind_k3        *2*ncunk)*nnzp] = tmp_b_1_re;
	      hb_re[index + (ind_k3       + (ind_j2+ncunk)*2*ncunk)*nnzp] = hb_re[index + (ind_j2+ncunk + ind_k3        *2*ncunk)*nnzp] = tmp_b_2_re;
	      hb_re[index + (ind_k3+ncunk + ind_j2        *2*ncunk)*nnzp] = hb_re[index + (ind_j2       + (ind_k3+ncunk)*2*ncunk)*nnzp] = tmp_b_3_re;
	      hb_re[index + (ind_k3+ncunk + (ind_j2+ncunk)*2*ncunk)*nnzp] = hb_re[index + (ind_j2+ncunk + (ind_k3+ncunk)*2*ncunk)*nnzp] = tmp_b_4_re;

	      hb_im[index + (ind_k3       + ind_j2        *2*ncunk)*nnzp] = hb_im[index + (ind_j2       + ind_k3        *2*ncunk)*nnzp] = tmp_b_1_im;
	      hb_im[index + (ind_k3       + (ind_j2+ncunk)*2*ncunk)*nnzp] = hb_im[index + (ind_j2+ncunk + ind_k3        *2*ncunk)*nnzp] = tmp_b_2_im;
	      hb_im[index + (ind_k3+ncunk + ind_j2        *2*ncunk)*nnzp] = hb_im[index + (ind_j2       + (ind_k3+ncunk)*2*ncunk)*nnzp] = tmp_b_3_im;
	      hb_im[index + (ind_k3+ncunk + (ind_j2+ncunk)*2*ncunk)*nnzp] = hb_im[index + (ind_j2+ncunk + (ind_k3+ncunk)*2*ncunk)*nnzp] = tmp_b_4_im;

	      
	    }  // for(int k=0;k<ncoils;k++)
	  
	  }  // for(int j=0;j<ncoils;j++)

          
	}  // for(int sp2=sp-1;sp>=0;sp2--)
	
      }  // if(sp>0)
      

      // update backward Q products
      multiply_2spinors( a_1_re  ,a_1_im  ,b_1_re  ,b_1_im   , 
			 atot_re ,atot_im ,btot_re ,btot_im  ,
			 &atot_re,&atot_im,&btot_re,&btot_im );
      
    }  // for(int sp=nspokes-1;sp>=0;sp++)


    a_re[index] = atot_re;
    a_im[index] = atot_im;

    b_re[index] = btot_re;
    b_im[index] = btot_im;
    

  }  // if(index<nnzp)

    
}







