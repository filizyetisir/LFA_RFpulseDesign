



#include <hip/hip_runtime.h>
#include <cmath>


#define TWO_PI_GAMMA (2.675128976384781e+08)




__device__ void multiply_2spinors(double a1_re,double a1_im,double b1_re,double b1_im,
				  double a2_re,double a2_im,double b2_re,double b2_im,
				  double *a3_re,double *a3_im,double *b3_re,double *b3_im)
{
  *a3_re = a2_re*a1_re - a2_im*a1_im - b2_re*b1_re - b2_im*b1_im;
  *a3_im = a2_im*a1_re + a2_re*a1_im + b2_im*b1_re - b2_re*b1_im;
  
  *b3_re = b2_re*a1_re - b2_im*a1_im + a2_re*b1_re + a2_im*b1_im;
  *b3_im = b2_im*a1_re + b2_re*a1_im - a2_im*b1_re + a2_re*b1_im;
}




__device__ void multiply_3spinors(double a1_re,double a1_im,double b1_re,double b1_im,
				  double a2_re,double a2_im,double b2_re,double b2_im,
				  double a3_re,double a3_im,double b3_re,double b3_im,
				  double *a4_re,double *a4_im,double *b4_re,double *b4_im)
{
  
  double tmp1_re = a2_re*a1_re - a2_im*a1_im - b2_re*b1_re - b2_im*b1_im;
  double tmp1_im = a2_im*a1_re + a2_re*a1_im + b2_im*b1_re - b2_re*b1_im;
  
  double tmp2_re = b2_re*a1_re - b2_im*a1_im + a2_re*b1_re + a2_im*b1_im;
  double tmp2_im = b2_im*a1_re + b2_re*a1_im - a2_im*b1_re + a2_re*b1_im;
  
  *a4_re = a3_re*tmp1_re - a3_im*tmp1_im - b3_re*tmp2_re - b3_im*tmp2_im;
  *a4_im = a3_im*tmp1_re + a3_re*tmp1_im + b3_im*tmp2_re - b3_re*tmp2_im;
  
  *b4_re = b3_re*tmp1_re - b3_im*tmp1_im + a3_re*tmp2_re + a3_im*tmp2_im;
  *b4_im = b3_im*tmp1_re + b3_re*tmp1_im - a3_im*tmp2_re + a3_re*tmp2_im;    
}




// CUDA kernel
__global__ void f0_LFA_cudaKernel(double *a_re,double *a_im,double *b_re,double *b_im,
				  double *datot_dre_re,double *datot_dre_im,double *datot_dim_re,double *datot_dim_im,
				  double *dbtot_dre_re,double *dbtot_dre_im,double *dbtot_dim_re,double *dbtot_dim_im,
				  double *a_forw_re,double *a_forw_im,double *b_forw_re,double *b_forw_im,
				  double *b1s_re,double *b1s_im,double *grad,
				  int *time_to_sinc_time,int *time_to_spoke,double *subpulse,
				  double *b1maps_re,double *b1maps_im,
				  int nnzp,int ntimes,int ncoils,int nspokes,double deltat,int comp_grad)
{

  int index=threadIdx.x + blockIdx.x * blockDim.x;  //  global thread index == voxel index


  double a2_re,a2_im,b2_re,b2_im;
  double n1,n2,n3,phi,cosphi,sinphi;
      

  if(index<nnzp){

    double atot_re=1.0;
    double atot_im=0.0;

    double btot_re=0.0;
    double btot_im=0.0;

    if( comp_grad==1 ){
      a_forw_re[index]=1.0;
      a_forw_im[index]=0.0;
      
      b_forw_re[index]=0.0;
      b_forw_im[index]=0.0;
    }

    
    // FORWARD BLOCH SIMULATION    
    for(int time=0;time<ntimes;time++){
      
      double norm=b1s_re[time*nnzp+index]*b1s_re[time*nnzp+index] + b1s_im[time*nnzp+index]*b1s_im[time*nnzp+index] + grad[time*nnzp+index]*grad[time*nnzp+index];
      norm=sqrt(norm);
      
      if(norm==0){
	
	a2_re=1.0;
	a2_im=0.0;
	
	b2_re=0.0;
	b2_im=0.0;	
	
      }else{
	
	phi=-deltat*norm*TWO_PI_GAMMA;
	
	n1=b1s_re[time*nnzp+index] / norm;
	n2=b1s_im[time*nnzp+index] / norm;
	n3=grad[time*nnzp+index] / norm;    
	
	// new CK parameters
	sincos( phi/2.0,&sinphi,&cosphi );
	
	a2_re=cosphi;
	a2_im=-n3*sinphi;
	
	b2_re=n2*sinphi;              
	b2_im=-n1*sinphi;              
	
	// forward Q product
	multiply_2spinors( atot_re,atot_im,btot_re,btot_im,
			   a2_re,a2_im,b2_re,b2_im,
			   &atot_re,&atot_im,&btot_re,&btot_im );
      }


      // if gradient is required, store one time late forward Q matrix product      
      if( comp_grad==1 && time<ntimes-1 ){
	  a_forw_re[index + (time+1)*nnzp]=atot_re;  // better way to handle memory?
	  a_forw_im[index + (time+1)*nnzp]=atot_im;  
	  
	  b_forw_re[index + (time+1)*nnzp]=btot_re;  
	  b_forw_im[index + (time+1)*nnzp]=btot_im;  	
      }

      
    }  // for(time=0;times<ntimes;time++)     
    

    a_re[index] = atot_re;
    a_im[index] = atot_im;

    b_re[index] = btot_re;
    b_im[index] = btot_im;   
    
    // BACKWARD BLOCH SIMULATION        
    if(comp_grad==1){

      atot_re = 1.0;
      atot_im = 0.0;

      btot_re = 0.0;
      btot_im = 0.0;

      for(int j=0;j<nspokes*ncoils;j++){
	datot_dre_re[index + j*nnzp] = 0.0;
	datot_dre_im[index + j*nnzp] = 0.0;
	datot_dim_re[index + j*nnzp] = 0.0;
	datot_dim_im[index + j*nnzp] = 0.0;

	dbtot_dre_re[index + j*nnzp] = 0.0;
	dbtot_dre_im[index + j*nnzp] = 0.0;
	dbtot_dim_re[index + j*nnzp] = 0.0;
	dbtot_dim_im[index + j*nnzp] = 0.0;
      }
      
      for(int time=ntimes-1;time>=0;time--){            
	
	int sp_num=time_to_spoke[time];
	int sinctime=time_to_sinc_time[time];

	// compute CK parameters
	double norm=b1s_re[time*nnzp+index]*b1s_re[time*nnzp+index] + b1s_im[time*nnzp+index]*b1s_im[time*nnzp+index] + grad[time*nnzp+index]*grad[time*nnzp+index];
	norm = sqrt(norm);

	if(norm==0){	  
	  a2_re=1.0;
	  a2_im=0.0;
	  
	  b2_re=0.0;
	  b2_im=0.0;	  
	}else{	  
	  phi=-deltat*norm*TWO_PI_GAMMA;
	  
	  n1=b1s_re[time*nnzp+index] / norm;
	  n2=b1s_im[time*nnzp+index] / norm;
	  n3=grad[time*nnzp+index] / norm;    
	  
	  sincos( phi/2.0,&sinphi,&cosphi );
	  
	  a2_re=cosphi;
	  a2_im=-n3*sinphi;
	  
	  b2_re=n2*sinphi;              
	  b2_im=-n1*sinphi;              	  
	}

	
	if( sp_num>0 && sinctime>0 && norm>0 ){               
	  
	  for(int j=0;j<ncoils;j++){
	      
	    // derivative of phi wrt spokes amplitudes
	    double dphi_dre = -deltat/norm*subpulse[sinctime-1]*TWO_PI_GAMMA * ( b1s_re[index+time*nnzp]*b1maps_re[index+j*nnzp] + b1s_im[index+time*nnzp]*b1maps_im[index+j*nnzp] );
	    double dphi_dim = deltat/norm*subpulse[sinctime-1]*TWO_PI_GAMMA * ( b1s_re[index+time*nnzp]*b1maps_im[index+j*nnzp] - b1s_im[index+time*nnzp]*b1maps_re[index+j*nnzp] );
	    
	    // derivative of n (rotation axis) wrt spokes amplitudes
	    double tmp_re=-subpulse[sinctime-1]/(norm*norm*norm) * ( b1s_re[index+time*nnzp]*b1maps_re[index+j*nnzp] + b1s_im[index+time*nnzp]*b1maps_im[index+j*nnzp] );
	    double tmp_im=-subpulse[sinctime-1]/(norm*norm*norm) * ( b1s_re[index+time*nnzp]*b1maps_im[index+j*nnzp] - b1s_im[index+time*nnzp]*b1maps_re[index+j*nnzp] );
	    
	    double tmp3=subpulse[sinctime-1]/norm;            
	    
	    double dnx_dre=tmp_re*b1s_re[index+time*nnzp]  + tmp3*b1maps_re[index+j*nnzp];
	    double dnx_dim=-tmp_im*b1s_re[index+time*nnzp] - tmp3*b1maps_im[index+j*nnzp];
	    
	    double dny_dre=tmp_re*b1s_im[index+time*nnzp] + tmp3*b1maps_im[index+j*nnzp];
	    double dny_dim=-tmp_im*b1s_im[index+time*nnzp] + tmp3*b1maps_re[index+j*nnzp];
	    
	    double dnz_dre=tmp_re * grad[index+time*nnzp];
	    double dnz_dim=-tmp_im * grad[index+time*nnzp];
	    
	    // derivative of Q 
	    double da_dre_re = -dphi_dre/2.0*sinphi;
	    double da_dre_im = -dphi_dre/2.0*n3*cosphi - sinphi*dnz_dre;
	    
	    double da_dim_re = -dphi_dim/2.0*sinphi;
	    double da_dim_im = -dphi_dim/2.0*n3*cosphi - sinphi*dnz_dim;            
	    
	    double db_dre_re =  dny_dre*sinphi + 0.5*n2*cosphi*dphi_dre;
	    double db_dre_im = -dnx_dre*sinphi - 0.5*n1*cosphi*dphi_dre;                                
	    
	    double db_dim_re =  dny_dim*sinphi + 0.5*n2*cosphi*dphi_dim;
	    double db_dim_im = -dnx_dim*sinphi - 0.5*n1*cosphi*dphi_dim;
	    
	    
	    // update derivatives of total Q matrices (sandwitch product)
	    double tmp3_re,tmp3_im,tmp4_re,tmp4_im;
	    multiply_3spinors( a_forw_re[index+time*nnzp],a_forw_im[index+time*nnzp],b_forw_re[index+time*nnzp],b_forw_im[index+time*nnzp],
			       da_dre_re                 ,da_dre_im                 ,db_dre_re                 ,db_dre_im                 ,
			       atot_re                   ,atot_im                   ,btot_re                   ,btot_im                   ,
			       &tmp3_re                  ,&tmp3_im                  ,&tmp4_re                  ,&tmp4_im                  );                           
	    	    
	    datot_dre_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils] += tmp3_re;
	    datot_dre_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils] += tmp3_im;
	    
	    dbtot_dre_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils] += tmp4_re;
	    dbtot_dre_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils] += tmp4_im;
            

	    multiply_3spinors( a_forw_re[index+time*nnzp],a_forw_im[index+time*nnzp],b_forw_re[index+time*nnzp],b_forw_im[index+time*nnzp],
			       da_dim_re                 ,da_dim_im                 ,db_dim_re                 ,db_dim_im                 ,
			       atot_re                   ,atot_im                   ,btot_re                   ,btot_im                   ,
			       &tmp3_re                  ,&tmp3_im                  ,&tmp4_re                  ,&tmp4_im                  );                           
	    
	    datot_dim_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils] += tmp3_re;
	    datot_dim_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils] += tmp3_im;
	    
	    dbtot_dim_re[index + j*nnzp + (sp_num-1)*nnzp*ncoils] += tmp4_re;            
	    dbtot_dim_im[index + j*nnzp + (sp_num-1)*nnzp*ncoils] += tmp4_im;
	    
	  }  // for(j=0;j<ncoils;j++)
          
	}  // if(sp_num>0)
	
	// keep track of the backward Q matrices product
	multiply_2spinors( a2_re   ,a2_im   ,b2_re   ,b2_im   ,
			   atot_re ,atot_im ,btot_re ,btot_im ,
			   &atot_re,&atot_im,&btot_re,&btot_im);
        
      }  // for(time=ntimes-1;time>=0;time--)
      
    }  // if(comp_grad==1)


  }  //  if(index<nnzp)

  
}




















