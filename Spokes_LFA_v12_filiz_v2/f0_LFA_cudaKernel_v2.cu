



#include <hip/hip_runtime.h>
#include <cmath>


#define TWO_PI_GAMMA (2.675128976384781e+08)




__device__ void multiply_2spinors(double a1_re,double a1_im,double b1_re,double b1_im,
				  double a2_re,double a2_im,double b2_re,double b2_im,
				  double *a3_re,double *a3_im,double *b3_re,double *b3_im)
{
  *a3_re = a2_re*a1_re - a2_im*a1_im - b2_re*b1_re - b2_im*b1_im;
  *a3_im = a2_im*a1_re + a2_re*a1_im + b2_im*b1_re - b2_re*b1_im;
  
  *b3_re = b2_re*a1_re - b2_im*a1_im + a2_re*b1_re + a2_im*b1_im;
  *b3_im = b2_im*a1_re + b2_re*a1_im - a2_im*b1_re + a2_re*b1_im;
}




__device__ void multiply_3spinors(double a1_re,double a1_im,double b1_re,double b1_im,
				  double a2_re,double a2_im,double b2_re,double b2_im,
				  double a3_re,double a3_im,double b3_re,double b3_im,
				  double *a4_re,double *a4_im,double *b4_re,double *b4_im)
{
  
  double tmp1_re = a2_re*a1_re - a2_im*a1_im - b2_re*b1_re - b2_im*b1_im;
  double tmp1_im = a2_im*a1_re + a2_re*a1_im + b2_im*b1_re - b2_re*b1_im;
  
  double tmp2_re = b2_re*a1_re - b2_im*a1_im + a2_re*b1_re + a2_im*b1_im;
  double tmp2_im = b2_im*a1_re + b2_re*a1_im - a2_im*b1_re + a2_re*b1_im;
  
  *a4_re = a3_re*tmp1_re - a3_im*tmp1_im - b3_re*tmp2_re - b3_im*tmp2_im;
  *a4_im = a3_im*tmp1_re + a3_re*tmp1_im + b3_im*tmp2_re - b3_re*tmp2_im;
  
  *b4_re = b3_re*tmp1_re - b3_im*tmp1_im + a3_re*tmp2_re + a3_im*tmp2_im;
  *b4_im = b3_im*tmp1_re + b3_re*tmp1_im - a3_im*tmp2_re + a3_re*tmp2_im;    
}




// CUDA kernel
__global__ void f0_LFA_cudaKernel(double *a_re,double *a_im,double *b_re,double *b_im,
				  double *b1s_re,double *b1s_im,double *grad,
				  double *time_to_sinc_time,double *time_to_spoke,double *subpulse,
				  double *b1maps_re,double *b1maps_im,double *target_LFA_re,double *target_LFA_im,
				  int nnzp,int ntimes,int ncoils,int nspokes,double deltat,int comp_grad)
{


  int index=threadIdx.x + blockIdx.x * blockDim.x;  //  global thread index == voxel index


  if(index<nnzp){

    a_re[0]=1.0;
    a_im[0]=0.0;

    b_re[0]=0.0;
    b_im[0]=0.0;

    
    // FORWARD BLOCH SIMULATION
    for(int time=0;time<ntimes;time++){
      
      double norm=b1s_re[time*nnzp+index]*b1s_re[time*nnzp+index] + b1s_im[time*nnzp+index]*b1s_im[time*nnzp+index] + grad[time*nnzp+index]*grad[time*nnzp+index];
      norm=sqrt(norm);
      
      double a2_re,a2_im,b2_re,b2_im;
      double n1,n2,n3,phi,cosphi,sinphi;
      if(norm==0){
	
	a2_re=1.0;
	a2_im=0.0;
	
	b2_re=0.0;
	b2_im=0.0;	
	
      }else{
	
	phi=-deltat*norm*TWO_PI_GAMMA;
	
	n1=b1s_re[time*nnzp+index] / norm;  // not good to read these from global memory!! IMPROVE!!
	n2=b1s_im[time*nnzp+index] / norm;
	n3=grad[time*nnzp+index] / norm;    
	
	// new CK parameters
	sincos( phi/2.0,&sinphi,&cosphi );
	
	a2_re=cosphi;
	a2_im=-n3*sinphi;
	
	b2_re=n2*sinphi;              
	b2_im=-n1*sinphi;              
	
	// forward Q product
	multiply_2spinors( a_re[index],a_im[index],b_re[index],b_im[index],
			   a2_re,a2_im,b2_re,b2_im,
			   &a_re[index],&a_im[index],&b_re[index],&b_im[index]);
      }      
      
    }  // for(time=0;times<ntimes;time++)     
    
  }  //  if(index<nnzp)
  
}




















